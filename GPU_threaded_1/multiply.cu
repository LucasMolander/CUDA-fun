#include "hip/hip_runtime.h"
/**
 * This ended up not being as efficient as CPU threads.
 * 
 * See the "Results 1" results for the benchmark data.
 */

#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>

#include <hip/hip_runtime.h>

#include "../matrix.c"
#include "../matrix.h"

void fail(const char *message)
{
    printf(message);
    exit(EXIT_FAILURE);
}

__global__ void doMultiply(int *d_result, int *d_a, int *d_b, 
    int m, int n, int p, int nThreads)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    int nElements = m * p;

    while (idx < nElements) {
        int r = idx / p; // Integer division on purpose
        int c = idx % p;

        int *v1 = d_a + (r * n);
        int *v2 = d_b + (c * n);

        int dotProd = 0;
        for (int i = 0; i < n; i++) {
            dotProd += v1[i] * v2[i];
        }

        d_result[r * p + c] = dotProd;

        idx += nThreads;
    }
}

/**
 * Assumes a is row-major and b is column-major.
 * 
 * Result is always row-major.
 */
Matrix *multiply(Matrix *a, Matrix *b)
{
    Matrix *result = generateMatrix(a->nRows, b->nCols, true);

    // Move A to device
    int *d_a = NULL;
    if (hipMalloc((void **) &d_a, a->nRows * a->nCols * sizeof(int)) != hipSuccess)
        fail("Failed to allocate space for A");
        
    if (hipMemcpy(d_a, a->values, a->nRows * a->nCols * sizeof(int), hipMemcpyHostToDevice) != hipSuccess)
        fail("Failed to copy A over");

    // Move B to device
    int *d_b = NULL;
    if (hipMalloc((void **) &d_b, b->nRows * b->nCols * sizeof(int)) != hipSuccess)
        fail("Failed to allocate space for B");
    
    if (hipMemcpy(d_b, b->values, b->nRows * b->nCols * sizeof(int), hipMemcpyHostToDevice) != hipSuccess)
        fail("Failed to copy B over");

    // Allocate space for AB
    int *d_result = NULL;
    if (hipMalloc((void **)&d_result, a->nRows * b->nCols * sizeof(int)) != hipSuccess)
        fail("Failed to allocate space for the result matrix");

    // Calculate
    // int threadsPerBlock = 192;
    // int nBlocks = 13;
    int threadsPerBlock = 128;
    int nBlocks = 32;
    int nThreads = threadsPerBlock * nBlocks;
    int nElements = a->nRows * b->nCols;
    
    doMultiply<<<nBlocks, threadsPerBlock>>>(d_result, d_a, d_b, a->nRows, a->nCols, b->nCols, nThreads);
    if (hipGetLastError() != hipSuccess)
        fail("Failure in CUDA kernel execution");
    
    if (hipMemcpy(result->values, d_result, nElements * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess)
        fail("Failed to copy result matrix to host");

    return result;
}

int main(int argc, char *argv[])
{
    // Ensure enough arguments exist
    if (argc < 5)
        fail("Required arguments: nRows1 nCols1 nRows2 nCols2\n");

    int nRows1,
        nCols1,
        nRows2,
        nCols2;

    // It's okay that atoi returns 0 on invalid
    // because 0 is an invalid matrix dimension
    if ((nRows1 = atoi(argv[1])) == 0)
        fail("Invalid matrix dimension.\n");
    if ((nCols1 = atoi(argv[2])) == 0)
        fail("Invalid matrix dimension.\n");
    if ((nRows2 = atoi(argv[3])) == 0)
        fail("Invalid matrix dimension.\n");
    if ((nCols2 = atoi(argv[4])) == 0)
        fail("Invalid matrix dimension.\n");

    // Negative matrix dimensions are also bad
    if (nRows1 < 0 || nCols1 < 0 || nRows2 < 0 || nCols2 < 0)
        fail("Invalid matrix dimension.\n");
    
    // Make sure the matrix multiplication is valid
    if (nCols1 != nRows2)
        fail("Matrices cannot be multiplied (nCols1 needs to equal nRows2)\n");

    // Echo matrix dimensions to the user
    // printf("%d x %d\n", nRows1, nCols1);
    // printf("%d x %d\n", nRows2, nCols2);
    // printf("\n");

    Matrix *a = generateMatrix(nRows1, nCols1, true);
    fillMatrixStepwise(a);
    
    Matrix *b = generateMatrix(nRows2, nCols2, false);
    fillMatrixStepwise(b);

    Matrix *ab = multiply(a, b);

    // printMatrix(a);
    // printf("\n");
    // printMatrix(b);
    // printf("\n");
    // printMatrix(ab);

    // Clean up
    destroyMatrix(a);
    destroyMatrix(b);
    destroyMatrix(ab);
    
    hipDeviceReset();

    return EXIT_SUCCESS;
}
